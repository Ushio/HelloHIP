
#include <hip/hip_runtime.h>
extern "C" __global__ void hoge( float* a, float *b ) 
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float bs[4];
    for( int i = 0 ; i < 4 ; ++i )
    {
        bs[i] = b[i];
    }
    for( int i = 0 ; i < 4 ; ++i )
    {
        a[idx] += bs[i];
    }
}